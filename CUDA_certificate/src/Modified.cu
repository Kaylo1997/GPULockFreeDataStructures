
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define block_size   1024

__global__ void calculation(    int *a, 
                                int *b, 
                                int *c, 
                                int constant, 
                                int vector_size ) {
        int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    // this thread handles the data at its thread id		
	__shared__ float myblock1[block_size+2];

	myblock1[tid] = a[tid];
	

	__syncthreads();

        if (tid < vector_size){
        
                // Read in inputs
                int prev_a = myblock1[tid>0?tid-1:(vector_size-1)];
                int curr_a = myblock1[tid];
                int post_a = myblock1[tid<(vector_size-1)?tid+1:0];
                
                int curr_b = b[tid];
                
                // Do computation
                int output_c = (prev_a-post_a)*curr_b + curr_a*constant;
                
                // Write result
                c[tid] = output_c;               
        }

	a[tid] = myblock1[tid];
}

int main( int argc, char* argv[] ) { 

        // Parse Input arguments
        
        // Check the number of arguments (we only receive command + vector size)
        if (argc != 2) {
                // Tell the user how to run the program
                printf ("Usage: %s vector_size\n", argv[0]);
                // "Usage messages" are a conventional way of telling the user
                // how to run a program if they enter the command incorrectly.
                return 1;
        }
        // Set GPU Variables based on input arguments
        int vector_size = atoi(argv[1]);
        int grid_size   = ((vector_size-1)/block_size) + 1;
                
        // Set device that we will use for our cuda code
        // It will be either 0 or 1
        hipSetDevice(0);
        
	// Time Variables
	hipEvent_t start_cpu, start_gpu;
        hipEvent_t  stop_cpu,  stop_gpu;
	
        hipEventCreate (&start_cpu);
	hipEventCreate (&start_gpu);
        
	hipEventCreate (&stop_cpu);
	hipEventCreate (&stop_gpu);
	
	float time;
        
        // Input Arrays and variables
        int *a          = new int [vector_size]; 
        int *b          = new int [vector_size]; 
        int *c_cpu      = new  int [vector_size]; 
        int *c_gpu      = new  int [vector_size];
        int constant    = 4;

        // Pointers in GPU memory
        int *dev_a;
        int *dev_b;
        int  *dev_c;

        // fill the arrays 'a' and 'b' on the CPU
	printf("Filling up input arrays with random values between 1 and 10.\n");
        for (int i = 0; i < vector_size; i++) {
                a[i] = rand()%10;
                b[i] = rand()%10;
        }

        //
        // CPU Calculation
        //////////////////
        
	printf("Running sequential job.\n");
	hipEventRecord(start_cpu,0);
        
        // Calculate C in the CPU
        for (int i = 0; i < vector_size; i++) {
                // Read in inputs
                int prev_a = a[i>0?i-1:(vector_size-1)];
                int curr_a = a[i];
                int post_a = a[i<(vector_size-1)?i+1:0];
                
                int curr_b = b[i];
                
                // Do computation
                int output_c = (prev_a-post_a)*curr_b + curr_a*constant;
                
                // Write result
                c_cpu[i] = output_c;
        }
        
	hipEventRecord(stop_cpu,0);
	hipEventSynchronize(stop_cpu);
        
	hipEventElapsedTime(&time, start_cpu, stop_cpu);
	printf("\tSequential Job Time: %.2f ms\n", time);
      
        //
        // GPU Calculation
        //////////////////
        
        printf("Running parallel job.\n");
        
	hipEventRecord(start_gpu,0);
        
        // allocate the memory on the GPU
        hipMalloc( (void**)&dev_a,       vector_size * sizeof(int) );
        hipMalloc( (void**)&dev_b,       vector_size * sizeof(int) );
        hipMalloc( (void**)&dev_c,       vector_size * sizeof(int) );

        // set arrays to 0
        hipMemset(dev_a,         0, vector_size * sizeof(int));
        hipMemset(dev_b,         0, vector_size * sizeof(int));
        hipMemset(dev_c,         0, vector_size * sizeof(int));
        
        // copy the arrays 'a' and 'b' to the GPU
        hipMemcpy( dev_a, a, vector_size * sizeof(int),
                              hipMemcpyHostToDevice );
        hipMemcpy( dev_b, b, vector_size * sizeof(int),
                              hipMemcpyHostToDevice );
        // run kernel
        calculation<<<grid_size,block_size>>>(  dev_a, 
                                                dev_b, 
                                                dev_c, 
                                                constant,
                                                vector_size );
                                                        
        // copy the array 'c' back from the GPU to the CPU
        hipMemcpy( c_gpu, dev_c, vector_size * sizeof(int),
                              hipMemcpyDeviceToHost );

	hipEventRecord(stop_gpu,0);
	hipEventSynchronize(stop_gpu);

	hipEventElapsedTime(&time, start_gpu, stop_gpu);
	printf("\tParallel Job Time: %.2f ms\n", time);

        // compare the results
        int error = 0;
        for (int i = 0; i < vector_size; i++) {
                if (c_cpu[i] != c_gpu[i]){
                        error = 1;
                        printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
                }
		if (error) break; 
        }
        
        if (error == 0){
                printf ("Correct result. No errors were found.\n");
        }

        // free the memory allocated on the GPU
        hipFree( dev_a );
        hipFree( dev_b );
        hipFree( dev_c );
        
        // free cuda events
        hipEventDestroy (start_cpu);
	hipEventDestroy (start_gpu);
        
	hipEventDestroy (stop_cpu);
	hipEventDestroy (stop_gpu);
        
        // free CPU memory        
	free(a);
	free(b);
	free(c_cpu);
	free(c_gpu);
	

       return 0;
}
